#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define ROWS 4096
#define COLS 4096

__global__ void histo(int* d_hist1, int* d_hist2, int* mat) {

	int id;
	id = blockIdx.x * blockDim.x + threadIdx.x;

	switch (d_hist1[id]) {
	case 0:
		atomicAdd(&mat[0], 1);
		break;
	case 1:
		atomicAdd(&mat[1], 1);
		break;
	case 2:
		atomicAdd(&mat[2], 1);
		break;
	case 3:
		atomicAdd(&mat[3], 1);
		break;
	case 4:
		atomicAdd(&mat[4], 1);
		break;
	case 5:
		atomicAdd(&mat[5], 1);
		break;
	case 6:
		atomicAdd(&mat[6], 1);
		break;
	case 7:
		atomicAdd(&mat[7], 1);
		break;
	case 8:
		atomicAdd(&mat[8], 1);
		break;
	case 9:
		atomicAdd(&mat[9], 1);
		break;

	}
	__syncthreads();
	switch (d_hist2[id]) {
	case 0:
		atomicAdd(&mat[0], 1);
		break;
	case 1:
		atomicAdd(&mat[1], 1);
		break;
	case 2:
		atomicAdd(&mat[2], 1);
		break;
	case 3:
		atomicAdd(&mat[3], 1);
		break;
	case 4:
		atomicAdd(&mat[4], 1);
		break;
	case 5:
		atomicAdd(&mat[5], 1);
		break;
	case 6:
		atomicAdd(&mat[6], 1);
		break;
	case 7:
		atomicAdd(&mat[7], 1);
		break;
	case 8:
		atomicAdd(&mat[8], 1);
		break;
	case 9:
		atomicAdd(&mat[9], 1);
		break;


	}

}

int main(int argc, char* argv[]) {

	//int histogram[ROWS][COLS];
	int x, y;
	int hist[ROWS * COLS], hist1[(ROWS * COLS) / 2], hist2[(ROWS * COLS) / 2];
	int *d_hist1, *d_hist2;
	int i = 0;
	int mat[10];
	int *d_mat;


	FILE *fp;
	fp = fopen("input.txt", "r");
	for (x = 0; x < ROWS; x++)
	{
		for ( y = 0; y < COLS; y++)
		{
			fscanf(fp, "%d", &hist[i]);
			// printf("%d\t",hist[i]);
			i++;
		}

	}
	fclose(fp);

	printf("\n");

	//int i = 0;
	/*for (x = 0; x < ROWS; x++) {
			for (y = 0; y < COLS; y++) {
					hist[i] = histogram[x][y];
			}
	}*/

	for (i = 0; i < (ROWS * COLS) / 2; i++) {
		hist1[i] = hist[i];
		//	printf("%d\t", hist[i]);
	}
	printf("\n");
	for (i = 0; i < (ROWS * COLS) / 2; i++) {
		hist2[i] = hist[((ROWS * COLS) / 2) + i];
		//	printf("%d\t", hist[i]);
	}

	for (i = 0; i < 10; i++) {
		mat[i] = 0;
	}

	hipMalloc(&d_hist1, ROWS * COLS * sizeof(int) / 2);
	hipMalloc(&d_hist2, ROWS * COLS * sizeof(int) / 2);
	hipMalloc(&d_mat, 10 * sizeof(int));


	hipMemcpy(d_hist1, hist1, (ROWS * COLS * sizeof(int)) / 2, hipMemcpyHostToDevice);
	hipMemcpy(d_hist2, hist2, (ROWS * COLS * sizeof(int)) / 2, hipMemcpyHostToDevice);
	hipMemcpy(d_mat, &mat, 10 * sizeof(int), hipMemcpyHostToDevice);
	hipMemset(d_mat, 0, sizeof(d_mat));


	//histo <<<16384,1024>>>(d_hist, d_mat, ROWS * COLS);
	histo <<< 8192, 1024>>>(d_hist1, d_hist2, d_mat);


	hipDeviceSynchronize();

	hipMemcpy(mat, d_mat, 10 * sizeof(int), hipMemcpyDeviceToHost);

	FILE *fout;
	fout = fopen("output.txt", "w");

	for (i = 0; i < 10; i++) {
		printf("\n%d => %d", i, mat[i]);
		fprintf(fout, "%d => %d\n", i, mat[i]);
	}

	fclose(fout);
	hipFree(d_mat);
	hipFree(d_hist1);
	hipFree(d_hist2);
	return 0;

}
